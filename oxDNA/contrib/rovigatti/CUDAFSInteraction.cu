#include "hip/hip_runtime.h"
/*
 * CUDAFSInteraction.cu
 *
 *  Created on: 22/feb/2013
 *      Author: lorenzo
 */

#include "CUDAFSInteraction.h"

#include "../Lists/CUDASimpleVerletList.h"
#include "../Lists/CUDANoList.h"

/* BEGIN CUDA */

/* System constants */
__constant__ int MD_N[1];
__constant__ int MD_n_forces[1];
__constant__ int MD_N_patches[2];
__constant__ bool MD_one_component[1];
__constant__ float MD_box_side[1];
__constant__ float MD_sqr_rcut[1];
__constant__ float MD_sqr_rep_rcut[1];
__constant__ float MD_sqr_patch_rcut[1];
__constant__ float MD_sigma_ss[1];
__constant__ float MD_rcut_ss[1];
__constant__ float MD_lambda[1];
__constant__ float MD_A_part[1], MD_B_part[1];
__constant__ float4 MD_base_patches[2][CUDA_MAX_FS_PATCHES];

#include "../cuda_utils/CUDA_lr_common.cuh"

template <typename number, typename number4>
__device__ number4 minimum_image(number4 &r_i, number4 &r_j) {
	number dx = r_j.x - r_i.x;
	number dy = r_j.y - r_i.y;
	number dz = r_j.z - r_i.z;

	dx -= floorf(dx/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dy -= floorf(dy/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dz -= floorf(dz/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];

	return make_number4<number, number4>(dx, dy, dz, (number) 0.f);
}

template <typename number, typename number4>
__device__ void _particle_particle_interaction(number4 &ppos, number4 &qpos, number4 &a1, number4 &a2, number4 &a3, number4 &b1, number4 &b2, number4 &b3, number4 &F, number4 &torque) {
	int ptype = get_particle_type<number, number4>(ppos);
	int qtype = get_particle_type<number, number4>(qpos);

	number4 r = minimum_image<number, number4>(ppos, qpos);
	number sqr_r = CUDA_DOT(r, r);
	if(sqr_r >= MD_sqr_rcut[0]) return;

	// centre-centre
	number ir2 = 1.f / sqr_r;
	number lj_part = ir2*ir2*ir2;
	number force_module = -24.f * (lj_part - 2.f*SQR(lj_part)) / sqr_r;
	if(sqr_r >= MD_sqr_rep_rcut[0]) force_module = 0.f;
	F.x -= r.x * force_module;
	F.y -= r.y * force_module;
	F.z -= r.z * force_module;

	// TODO may be improved by removing branching
	if(ptype == qtype && !MD_one_component[0]) return;

	for(int pi = 0; pi < MD_N_patches[ptype]; pi++) {
		number4 ppatch = {
			a1.x*MD_base_patches[ptype][pi].x + a2.x*MD_base_patches[ptype][pi].y + a3.x*MD_base_patches[ptype][pi].z,
			a1.y*MD_base_patches[ptype][pi].x + a2.y*MD_base_patches[ptype][pi].y + a3.y*MD_base_patches[ptype][pi].z,
			a1.z*MD_base_patches[ptype][pi].x + a2.z*MD_base_patches[ptype][pi].y + a3.z*MD_base_patches[ptype][pi].z,
			0
		};

		for(int pj = 0; pj < MD_N_patches[qtype]; pj++) {
			number4 qpatch = {
				b1.x*MD_base_patches[qtype][pj].x + b2.x*MD_base_patches[qtype][pj].y + b3.x*MD_base_patches[qtype][pj].z,
				b1.y*MD_base_patches[qtype][pj].x + b2.y*MD_base_patches[qtype][pj].y + b3.y*MD_base_patches[qtype][pj].z,
				b1.z*MD_base_patches[qtype][pj].x + b2.z*MD_base_patches[qtype][pj].y + b3.z*MD_base_patches[qtype][pj].z,
				0
			};

			number4 patch_dist = {
				r.x + qpatch.x - ppatch.x,
				r.y + qpatch.y - ppatch.y,
				r.z + qpatch.z - ppatch.z,
				0
			};

			number dist = CUDA_DOT(patch_dist, patch_dist);
			if(dist < MD_sqr_patch_rcut[0]) {
				number r_p = sqrtf(dist);
				number exp_part = expf(MD_sigma_ss[0] / (r_p - MD_rcut_ss[0]));
				number energy_part = MD_A_part[0] * exp_part * (MD_B_part[0]/SQR(dist) - 1.);

				number force_mod  = MD_A_part[0] * exp_part * (4.*MD_B_part[0]/(SQR(dist)*r_p)) + MD_sigma_ss[0] * energy_part / SQR(r_p - MD_rcut_ss[0]);
				number4 tmp_force = patch_dist * (force_mod / r_p);

				torque -= _cross<number, number4>(ppatch, tmp_force);
				F.x -= tmp_force.x;
				F.y -= tmp_force.y;
				F.z -= tmp_force.z;
			}
		}
	}
}

// forces + second step without lists
template <typename number, typename number4>
__global__ void FS_forces(number4 *poss, GPU_quat<number> *orientations, number4 *forces, number4 *torques) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	number4 T = make_number4<number, number4>(0, 0, 0, 0);
	number4 ppos = poss[IND];
	GPU_quat<number> po = orientations[IND];
	number4 a1, a2, a3, b1, b2, b3;
	get_vectors_from_quat<number,number4>(po, a1, a2, a3);

	for(int j = 0; j < MD_N[0]; j++) {
		if(j != IND) {
			number4 qpos = poss[j];
			GPU_quat<number> qo = orientations[j];
			get_vectors_from_quat<number,number4>(qo, b1, b2, b3);
			_particle_particle_interaction<number, number4>(ppos, qpos, a1, a2, a3, b1, b2, b3, F, T);
		}
	}

	T = _vectors_transpose_number4_product(a1, a2, a3, T);

	forces[IND] = F;
	torques[IND] = T;
}

template <typename number, typename number4>
__global__ void FS_forces_edge(number4 *poss, GPU_quat<number> *orientations, number4 *forces, number4 *torques, edge_bond *edge_list,  int n_edges) {
	if(IND >= n_edges) return;

	number4 dF = make_number4<number, number4>(0, 0, 0, 0);
	number4 dT = make_number4<number, number4>(0, 0, 0, 0);

	edge_bond b = edge_list[IND];

	// get info for particle 1
	number4 ppos = poss[b.from];
	GPU_quat<number> po = orientations[b.from];

	// get info for particle 2
	number4 qpos = poss[b.to];
	GPU_quat<number> qo = orientations[b.to];

	number4 a1, a2, a3, b1, b2, b3;
	get_vectors_from_quat<number,number4>(po, a1, a2, a3);
	get_vectors_from_quat<number,number4>(qo, b1, b2, b3);

	_particle_particle_interaction<number, number4>(ppos, qpos, a1, a2, a3, b1, b2, b3, dF, dT);

	int from_index = MD_N[0]*(IND % MD_n_forces[0]) + b.from;
	if((dF.x*dF.x + dF.y*dF.y + dF.z*dF.z) > (number)0.f) LR_atomicAddXYZ(&(forces[from_index]), dF);
	if((dT.x*dT.x + dT.y*dT.y + dT.z*dT.z) > (number)0.f) LR_atomicAddXYZ(&(torques[from_index]), _vectors_transpose_number4_product(a1, a2, a3, dT));

	// Allen Eq. 6 pag 3:
	number4 dr = minimum_image<number, number4>(ppos, qpos); // returns qpos-ppos
	number4 crx = _cross<number, number4>(dr, dF);
	dT.x = -dT.x + crx.x;
	dT.y = -dT.y + crx.y;
	dT.z = -dT.z + crx.z;

	dF.x = -dF.x;
	dF.y = -dF.y;
	dF.z = -dF.z;

	int to_index = MD_N[0]*(IND % MD_n_forces[0]) + b.to;
	if((dF.x*dF.x + dF.y*dF.y + dF.z*dF.z) > (number)0.f) LR_atomicAddXYZ(&(forces[to_index]), dF);
	if((dT.x*dT.x + dT.y*dT.y + dT.z*dT.z) > (number)0.f) LR_atomicAddXYZ(&(torques[to_index]), _vectors_transpose_number4_product(b1, b2, b3, dT));
}

//Forces + second step with verlet lists
template <typename number, typename number4>
__global__ void FS_forces(number4 *poss, GPU_quat<number> *orientations, number4 *forces, number4 *torques, int *matrix_neighs, int *number_neighs) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	number4 T = make_number4<number, number4>(0, 0, 0, 0);
	number4 ppos = poss[IND];
	GPU_quat<number> po = orientations[IND];
	number4 a1, a2, a3, b1, b2, b3;
	get_vectors_from_quat<number,number4>(po, a1, a2, a3);

	int num_neighs = number_neighs[IND];

	for(int j = 0; j < num_neighs; j++) {
		int k_index = matrix_neighs[j*MD_N[0] + IND];

		number4 qpos = poss[k_index];
		GPU_quat<number> qo = orientations[k_index];
		get_vectors_from_quat<number,number4>(qo, b1, b2, b3);
		_particle_particle_interaction<number, number4>(ppos, qpos, a1, a2, a3, b1, b2, b3, F, T);
	}

	T = _vectors_transpose_number4_product(a1, a2, a3, T);

	forces[IND] = F;
	torques[IND] = T;
}

/* END CUDA PART */

#define HALF_ISQRT3 0.28867513459481292f

template<typename number, typename number4>
CUDAFSInteraction<number, number4>::CUDAFSInteraction() : CUDABaseInteraction<number, number4>(), FSInteraction<number>() {

}

template<typename number, typename number4>
CUDAFSInteraction<number, number4>::~CUDAFSInteraction() {

}

template<typename number, typename number4>
void CUDAFSInteraction<number, number4>::get_settings(input_file &inp) {
	FSInteraction<number>::get_settings(inp);
}

template<typename number, typename number4>
void CUDAFSInteraction<number, number4>::cuda_init(number box_side, int N) {
	CUDABaseInteraction<number, number4>::cuda_init(box_side, N);
	FSInteraction<number>::init();

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_one_component), &this->_one_component, sizeof(bool)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patches), &this->_N_patches, sizeof(int)) );
	if(!this->_one_component) CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patches), &this->_N_patches_B, sizeof(int), sizeof(int)) );

	float f_copy = box_side;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_box_side), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_rcut), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_rep_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_rep_rcut), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_patch_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_patch_rcut), &f_copy, sizeof(float)) );
	f_copy = this->_sigma_ss;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sigma_ss), &f_copy, sizeof(float)) );
	f_copy = this->_rcut_ss;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_rcut_ss), &f_copy, sizeof(float)) );
	f_copy = this->_lambda;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_lambda), &f_copy, sizeof(float)) );
	f_copy = this->_A_part;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_A_part), &f_copy, sizeof(float)) );
	f_copy = this->_B_part;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_B_part), &f_copy, sizeof(float)) );

	float4 base_patches[CUDA_MAX_FS_PATCHES];

	// ugly...
	int limit = (this->_one_component) ? 1 : 2;
	int n_patches = this->_N_patches;
	for(int i = 0; i < limit; i++) {
		switch(n_patches) {
		case 2: {
			base_patches[0] = make_float4(0, 0.5, 0, 0);
			base_patches[1] = make_float4(0, -0.5, 0, 0);
			break;
		}
		case 3: {
			number cos120 = cos(2 * M_PI / 3.);
			number sin120 = sin(2 * M_PI / 3.);

			base_patches[0] = make_float4(0, 1, 0, 0);
			base_patches[1] = make_float4(cos120, -sin120, 0, 0);
			base_patches[2] = make_float4(-cos120, -sin120, 0, 0);
			break;
		}
		case 4: {
			base_patches[0] = make_float4(-HALF_ISQRT3, -HALF_ISQRT3,  HALF_ISQRT3, 0);
			base_patches[1] = make_float4( HALF_ISQRT3, -HALF_ISQRT3, -HALF_ISQRT3, 0);
			base_patches[2] = make_float4( HALF_ISQRT3,  HALF_ISQRT3,  HALF_ISQRT3, 0);
			base_patches[3] = make_float4(-HALF_ISQRT3,  HALF_ISQRT3, -HALF_ISQRT3, 0);
			break;
		}
		default:
			throw oxDNAException("Unsupported number of patches %d", n_patches);
		}

		for(int j = 0; j < n_patches; j++) {
			number factor = 0.5 / sqrt(CUDA_DOT(base_patches[j], base_patches[j]));
			base_patches[j].x *= factor;
			base_patches[j].y *= factor;
			base_patches[j].z *= factor;
		}

		// fourth argument is the offset
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patches), base_patches, sizeof(float4)*n_patches, i*sizeof(float4)*CUDA_MAX_FS_PATCHES) );
		n_patches = this->_N_patches_B;
	}

	if(this->_N_patches > CUDA_MAX_FS_PATCHES) throw oxDNAException("CUDA supports only particles with up to %d patches", CUDA_MAX_FS_PATCHES);
	if(this->_use_edge) CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_n_forces), &this->_n_forces, sizeof(int)) );
}

template<typename number, typename number4>
void CUDAFSInteraction<number, number4>::compute_forces(CUDABaseList<number, number4> *lists, number4 *d_poss, GPU_quat<number> *d_orientations, number4 *d_forces, number4 *d_torques, LR_bonds *d_bonds) {
	CUDASimpleVerletList<number, number4> *_v_lists = dynamic_cast<CUDASimpleVerletList<number, number4> *>(lists);
	if(_v_lists != NULL) {
		if(_v_lists->use_edge()) {
				FS_forces_edge<number, number4>
					<<<(_v_lists->_N_edges - 1)/(this->_launch_cfg.threads_per_block) + 1, this->_launch_cfg.threads_per_block>>>
					//(d_poss, d_orientations, d_forces, d_torques, _v_lists->_d_edge_list, _v_lists->_N_edges);
					(d_poss, d_orientations, this->_d_edge_forces, this->_d_edge_torques, _v_lists->_d_edge_list, _v_lists->_N_edges);
				CUT_CHECK_ERROR("forces_second_step FS forces_edge");

				this->_sum_edge_forces_torques(d_forces, d_torques);
			}
			else {
				FS_forces<number, number4>
					<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
					(d_poss, d_orientations, d_forces, d_torques, _v_lists->_d_matrix_neighs, _v_lists->_d_number_neighs);
				CUT_CHECK_ERROR("forces_second_step FS simple_lists error");
			}
	}

	CUDANoList<number, number4> *_no_lists = dynamic_cast<CUDANoList<number, number4> *>(lists);
	if(_no_lists != NULL) {
		FS_forces<number, number4>
			<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
			(d_poss, d_orientations, d_forces, d_torques);
		CUT_CHECK_ERROR("forces_second_step FS no_lists error");
	}
}

template class CUDAFSInteraction<float, float4>;
template class CUDAFSInteraction<double, LR_double4>;
